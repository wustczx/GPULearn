#include "hip/hip_runtime.h"
#include<iostream>
#include"../common/base.h"

__global__ void add(int a, int b, int* c)
{
	*c = a+b;
}

int main()
{
	int c;
	int *dev_c;
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int)));
	add<<<1,1>>>(1, 2, dev_c);
	HANDLE_ERROR( hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
	std::cout<<"1+2="<<c<<std::endl;
	hipFree(dev_c);
	return 0;
}
